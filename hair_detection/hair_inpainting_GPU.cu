#include "hip/hip_runtime.h"
#include "hair_inpainting_GPU.cuh"

#define DATA_TILE_DIM 34

__constant__ float dt[1];
__constant__ float center_w[1];

__global__ void PDEHeatDiffusionSMEM(float* mask, float* src, float* tempSrc, int width, int height) {
	__shared__ float smem[DATA_TILE_DIM][DATA_TILE_DIM][3];
	int x = threadIdx.x + blockDim.x * blockIdx.x;
	int y = threadIdx.y + blockDim.y * blockIdx.y;
	if (x >= width || y >= height) return;
	int c1i = y * width + x;
	int c3i = y * (width * 3) + (x * 3);
	int c3ui = (y - 1) * (width * 3) + (x * 3);
	int c3di = (y + 1) * (width * 3) + (x * 3);
	int c3li = y * (width * 3) + ((x - 1) * 3);
	int c3ri = y * (width * 3) + ((x + 1) * 3);

	smem[threadIdx.y + 1][threadIdx.x + 1][0] = tempSrc[c3i]; // put your pixel
	smem[threadIdx.y + 1][threadIdx.x + 1][1] = tempSrc[c3i + 1]; // put your pixel
	smem[threadIdx.y + 1][threadIdx.x + 1][2] = tempSrc[c3i + 2]; // put your pixel

	// serious branching and bank confict
	if (x != 0 && threadIdx.x == 0) {
		smem[threadIdx.y + 1][threadIdx.x][0] = tempSrc[c3li]; // put your left pixel
		smem[threadIdx.y + 1][threadIdx.x][1] = tempSrc[c3li + 1]; // put your left pixel
		smem[threadIdx.y + 1][threadIdx.x][2] = tempSrc[c3li + 2]; // put your left pixel
	}
	if (x != width - 1 && threadIdx.x == TILE_DIM - 1) {
		smem[threadIdx.y + 1][threadIdx.x + 2][0] = tempSrc[c3ri]; // put your right pixel
		smem[threadIdx.y + 1][threadIdx.x + 2][1] = tempSrc[c3ri + 1]; // put your right pixel
		smem[threadIdx.y + 1][threadIdx.x + 2][2] = tempSrc[c3ri + 2]; // put your right pixel
	}
	if (y != 0 && threadIdx.y == 0) {
		smem[threadIdx.y][threadIdx.x + 1][0] = tempSrc[c3ui]; // put your up pixel
		smem[threadIdx.y][threadIdx.x + 1][1] = tempSrc[c3ui + 1]; // put your up pixel
		smem[threadIdx.y][threadIdx.x + 1][2] = tempSrc[c3ui + 2]; // put your up pixel
	}
	if (y != height - 1 && threadIdx.y == TILE_DIM - 1) {
		smem[threadIdx.y + 2][threadIdx.x + 1][0] = tempSrc[c3di]; // put your down pixel
		smem[threadIdx.y + 2][threadIdx.x + 1][1] = tempSrc[c3di + 1]; // put your down pixel
		smem[threadIdx.y + 2][threadIdx.x + 1][2] = tempSrc[c3di + 2]; // put your down pixel
	}
	__syncthreads();

	float center = smem[threadIdx.y + 1][threadIdx.x + 1][0];
	float mask_center = mask[c1i];
	tempSrc[c3i] = center
		+ dt[0] * (smem[threadIdx.y][threadIdx.x + 1][0] + smem[threadIdx.y + 2][threadIdx.x + 1][0] + smem[threadIdx.y + 1][threadIdx.x][0] + smem[threadIdx.y + 1][threadIdx.x + 2][0] - center_w[0] * center)
		- dt[0] * mask_center * (center - src[c3i]);

	center = smem[threadIdx.y + 1][threadIdx.x + 1][1];
	tempSrc[c3i + 1] = center
		+ dt[0] * (smem[threadIdx.y][threadIdx.x + 1][1] + smem[threadIdx.y + 2][threadIdx.x + 1][1] + smem[threadIdx.y + 1][threadIdx.x][1] + smem[threadIdx.y + 1][threadIdx.x + 2][1] - center_w[0] * center)
		- dt[0] * mask_center * (center - src[c3i + 1]);

	center = smem[threadIdx.y + 1][threadIdx.x + 1][2];
	tempSrc[c3i + 2] = center
		+ dt[0] * (smem[threadIdx.y][threadIdx.x + 1][2] + smem[threadIdx.y + 2][threadIdx.x + 1][2] + smem[threadIdx.y + 1][threadIdx.x][2] + smem[threadIdx.y + 1][threadIdx.x + 2][2] - center_w[0] * center)
		- dt[0] * mask_center * (center - src[c3i + 2]);
}

__global__ void PDEHeatDiffusion(float* mask, float* src, float* tempSrc, int width, int height, int ch) {
	int x = threadIdx.x + blockDim.x * blockIdx.x;
	int y = threadIdx.y + blockDim.y * blockIdx.y;
	if (x < 1 || y < 1 || x >= width -1 || y >= height - 1) return;
	float center;
	int c3i = y * (width * ch) + (x * ch);
	float mask_center = mask[y * width + x];

	for (int k = 0; k < ch; k++) {
		center = tempSrc[c3i+k];
		tempSrc[c3i+k] = center
			+ dt[0] * (tempSrc[(y - 1) * (width * ch) + (x * ch) + k]
				+ tempSrc[(y + 1) * (width * ch) + (x * ch) + k]
				+ tempSrc[y * (width * ch) + ((x - 1) * ch) + k]
				+ tempSrc[y * (width * ch) + ((x + 1) * ch) + k]
				- center_w[0] * center)
			- dt[0] * mask_center * (center - src[c3i+k]);
	}
}

void hairInpaintingGPU(cv::Mat& src, cv::Mat& mask, cv::Mat& dst, HairInpaintInfo info) {
	cv::resize(src, src, cv::Size(info.Width, info.Height));
	cv::resize(mask, mask, cv::Size(info.Width, info.Height));

#if DEBUG
	uchar* h_src = (uchar*)malloc(info.NumberOfC3Elements * sizeof(uchar));
	gpuErrorCheck(hipMemcpy(h_src, dSrc, info.NumberOfC3Elements * sizeof(uchar), hipMemcpyDeviceToHost));
	cv::Mat plot_src(info.Height, info.Width, CV_8UC3, h_src);
	displayImage(plot_src, "d_src", true);
#endif
	float* normalized_src = (float*)malloc(info.NumberOfC3Elements * sizeof(float));
	float* normalized_mask = (float*)malloc(info.NumberOfC1Elements * sizeof(float));
	float* normalized_masked_src = (float*)malloc(info.NumberOfC3Elements * sizeof(float));
	normalizeImage(src, mask, normalized_src, normalized_mask, normalized_masked_src, false);

#if DEBUG
	cv::Mat plot_normalized_src(info.Height, info.Width, CV_32FC3, normalized_src);
	cv::Mat plot_normalized_mask(info.Height, info.Width, CV_32FC1, normalized_mask);
	cv::Mat plot_normalized_masked_src(info.Height, info.Width, CV_32FC3, normalized_masked_src);
	displayImage(plot_normalized_src, "normalized_src", true);
	displayImage(plot_normalized_mask, "normalized_mask", true);
	displayImage(plot_normalized_masked_src, "normalized_masked_src", true);
#endif

	float* d_normalized_mask;
	float* d_normalized_masked_src;
	float* d_normalized_masked_src_updated;
	float* d_normalized_masked_src_temp;
	gpuErrorCheck(hipMalloc((float**)&d_normalized_mask, info.NumberOfC1Elements * sizeof(float)));
	gpuErrorCheck(hipMalloc((float**)&d_normalized_masked_src, info.NumberOfC3Elements * sizeof(float)));
	gpuErrorCheck(hipMalloc((float**)&d_normalized_masked_src_updated, info.NumberOfC3Elements * sizeof(float)));
	gpuErrorCheck(hipMalloc((float**)&d_normalized_masked_src_temp, info.NumberOfC3Elements * sizeof(float)));
	gpuErrorCheck(hipMemcpy(d_normalized_mask, normalized_mask, info.NumberOfC1Elements * sizeof(float), hipMemcpyHostToDevice));
	gpuErrorCheck(hipMemcpy(d_normalized_masked_src, normalized_masked_src, info.NumberOfC3Elements * sizeof(float), hipMemcpyHostToDevice));

	const int smem_size = DATA_TILE_DIM * DATA_TILE_DIM * info.Channels * sizeof(float);
	dim3 block(TILE_DIM, TILE_DIM);
	dim3 grid(iDivUp(info.Width, TILE_DIM), iDivUp(info.Height, TILE_DIM));

	const float h_const_dt = 0.1f;
	const float h_center_w = 4.0f;
	gpuErrorCheck(hipMemcpyToSymbol(HIP_SYMBOL(dt), &h_const_dt, 1 * sizeof(float)));
	gpuErrorCheck(hipMemcpyToSymbol(HIP_SYMBOL(center_w), &h_center_w, 1 * sizeof(float)));
	gpuErrorCheck(hipMemcpy(d_normalized_masked_src_temp, d_normalized_masked_src, info.NumberOfC3Elements * sizeof(float), hipMemcpyDeviceToDevice));

	for (int i = 0; i < info.Iters; i++) {
		PDEHeatDiffusion << <grid, block >> > (d_normalized_mask, d_normalized_masked_src, d_normalized_masked_src_temp, info.Width, info.Height, info.Channels);
		gpuErrorCheck(hipDeviceSynchronize());
	}

#if DEBUG
	float* h_result = (float*)malloc(info.NumberOfC3Elements * sizeof(float));
	gpuErrorCheck(hipMemcpy(h_result, d_normalized_masked_src_temp, info.NumberOfC3Elements * sizeof(float), hipMemcpyDeviceToHost));
	cv::Mat plot_src(info.Height, info.Width, CV_32FC3, h_result);
	cv::resize(plot_src, plot_src, cv::Size(info.Width * info.Rescale, info.Height * info.Rescale));
	displayImage(plot_src, "h_result", true);
#endif

	float* h_result = (float*)malloc(info.NumberOfC3Elements * sizeof(float));
	gpuErrorCheck(hipMemcpy(h_result, d_normalized_masked_src_temp, info.NumberOfC3Elements * sizeof(float), hipMemcpyDeviceToHost));
	cv::Mat dst_mat(info.Height, info.Width, CV_32FC3, h_result);
	cv::resize(dst_mat, dst_mat, cv::Size(info.Width * info.Rescale, info.Height * info.Rescale));
	dst = dst_mat;
	gpuErrorCheck(hipDeviceReset());
}

// main
void hairInpainting(cv::Mat& src, cv::Mat& mask, cv::Mat& dst, HairInpaintInfo info) {

}