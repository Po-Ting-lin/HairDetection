#include "hip/hip_runtime.h"
#include "hair_detection_kernel.cuh"
#include "utils.h"
#include <cuFFT.h>

#define TILE_DIM 32
#define BLOCK_DIM 8
#define EPSILON 1e-8
#define NUM_STREAMS 6
#define TIMER false
#define DEBUG false

__global__ void extractLChannelWithInstrinicFunction(uchar* src, float* dst, int nx, int ny, int nz) {
    int x = threadIdx.x + TILE_DIM * blockIdx.x;
    int y = threadIdx.y + TILE_DIM * blockIdx.y;

    for (int i = 0; i < TILE_DIM; i += BLOCK_DIM) {
        // take pixel from DRAM
        uchar R = *(src + ((y + i) * nx * nz) + (x * nz) + 0);
        uchar G = *(src + ((y + i) * nx * nz) + (x * nz) + 1);
        uchar B = *(src + ((y + i) * nx * nz) + (x * nz) + 2);

        // RGB to XYZ
        float r = fdividef((float)R, 255.0f);
        float g = fdividef((float)G, 255.0f);
        float b = fdividef((float)B, 255.0f);
        r = ((r > 0.04045f) ? __powf(fdividef(r + 0.055f, 1.055f), 2.4f) : fdividef(r, 12.92f)) * 100.0f;
        g = ((g > 0.04045f) ? __powf(fdividef(g + 0.055f, 1.055f), 2.4f) : fdividef(g, 12.92f)) * 100.0f;
        b = ((b > 0.04045f) ? __powf(fdividef(b + 0.055f, 1.055f), 2.4f) : fdividef(b, 12.92f)) * 100.0f;

        // XYZ to LAB
        float Y = fdividef(0.2126f * r + 0.7152f * g + 0.0722f * b, 100.0f);
        Y = (Y > 0.008856f) ? cbrtf(Y) : fmaf(7.787f, Y, 0.1379f);
        float L = fmaf(116.0f, Y, -16.0f) * 2.55f;

        // set pixel to DRAM
        *(dst + (y + i) * nx + x) = L;
    }
}

void getHairMask(cv::Mat& src, cv::Mat& dst, HairDetectionParameters para) {

#if TIMER
    auto t1 = std::chrono::system_clock::now();
#endif

    // declare 
    float
        * d_PaddedData,
        * d_Kernel,
        * d_PaddedKernel,
        * d_DepthResult;
    uchar
        * d_Result;

    fComplex
        * d_DataSpectrum,
        * d_KernelSpectrum,
        * d_TempSpectrum;

    hipfftHandle
        fftPlanFwd,
        fftPlanInv;

    uchar* src_ptr = src.data;
    const int dataH = src.rows;
    const int dataW = src.cols;
    const int depth = para.numberOfFilter;
    const int fftH = snapTransformSize(dataH + para.kernelH - 1);
    const int fftW = snapTransformSize(dataW + para.kernelW - 1);
    const unsigned long src_size = src.cols * src.rows * src.channels();
    const unsigned long src_byte_size = src_size * sizeof(uchar);
    const unsigned long src_c_size = src.cols * src.rows;
    const unsigned long src_c_byte_size = src_c_size * sizeof(float);

    // host data
    hipHostRegister(src_ptr, src_byte_size, hipHostRegisterDefault);

#if TIMER
    auto t2 = std::chrono::system_clock::now();
#endif

    // device data
    uchar* device_src_ptr;
    float* device_src_c_ptr;
    gpuErrorCheck(hipMalloc((uchar**)&device_src_ptr, src_byte_size));
    gpuErrorCheck(hipMalloc((float**)&device_src_c_ptr, src_c_byte_size));

    // stream
    int SRC_DATA_PER_STREAM = src_size / NUM_STREAMS;
    int DST_DATA_PER_STREAM = src_c_size / NUM_STREAMS;
    int SRC_BYTES_PER_STREAM = src_byte_size / NUM_STREAMS;
    int DST_BYTES_PER_STREAM = src_c_byte_size / NUM_STREAMS;

    hipStream_t stream[NUM_STREAMS];
    for (int i = 0; i < NUM_STREAMS; i++) {
        hipStreamCreate(&stream[i]);
    }

    int block_x_size = TILE_DIM;
    int block_y_size = BLOCK_DIM;
    int grid_x_size = (src.cols + TILE_DIM - 1) / TILE_DIM;
    int pruned_rows = src.rows / NUM_STREAMS;
    int grid_y_size = (pruned_rows + TILE_DIM - 1) / TILE_DIM;

    dim3 block(block_x_size, block_y_size);
    dim3 grid(grid_x_size, grid_y_size);

    int src_offset = 0;
    int dst_offset = 0;

    for (int i = 0; i < NUM_STREAMS; i++) {
        src_offset = i * SRC_DATA_PER_STREAM;
        dst_offset = i * DST_DATA_PER_STREAM;
        gpuErrorCheck(hipMemcpyAsync(&device_src_ptr[src_offset], &src_ptr[src_offset], SRC_BYTES_PER_STREAM, hipMemcpyHostToDevice, stream[i]));
        extractLChannelWithInstrinicFunction << < grid, block, 0, stream[i] >> > (&device_src_ptr[src_offset], &device_src_c_ptr[dst_offset], src.cols, pruned_rows, src.channels());
    }

#if TIMER
    auto t3 = std::chrono::system_clock::now();
#endif

    gpuErrorCheck(hipDeviceSynchronize());

    for (int i = 0; i < NUM_STREAMS; i++) {
        hipStreamDestroy(stream[i]);
    }

    hipHostUnregister(src_ptr);
    gpuErrorCheck(hipFree(device_src_ptr));

#if TIMER
    auto t4 = std::chrono::system_clock::now();
#endif

    // init data
    float* h_kernels = gaborFilterCube(para);

#if TIMER
    auto t5 = std::chrono::system_clock::now();
#endif

    // allocation
    gpuErrorCheck(hipMalloc((void**)&d_Kernel, para.kernelH * para.kernelW * para.numberOfFilter * sizeof(float)));

    gpuErrorCheck(hipMalloc((void**)&d_PaddedData, fftH * fftW * sizeof(float)));
    gpuErrorCheck(hipMalloc((void**)&d_PaddedKernel, fftH * fftW * sizeof(float)));
    gpuErrorCheck(hipMalloc((void**)&d_DepthResult, fftH * fftW * para.numberOfFilter * sizeof(float)));
    gpuErrorCheck(hipMalloc((void**)&d_Result, dataH * dataW * sizeof(uchar)));

    gpuErrorCheck(hipMalloc((void**)&d_DataSpectrum, fftH * (fftW / 2 + 1) * sizeof(fComplex)));
    gpuErrorCheck(hipMalloc((void**)&d_KernelSpectrum, fftH * (fftW / 2 + 1) * sizeof(fComplex)));
    gpuErrorCheck(hipMalloc((void**)&d_TempSpectrum, fftH * (fftW / 2 + 1) * sizeof(fComplex)));

    // H to D
    gpuErrorCheck(hipMemcpy(d_Kernel, h_kernels, para.kernelH * para.kernelW * para.numberOfFilter * sizeof(float), hipMemcpyHostToDevice));

#if TIMER
    auto t6 = std::chrono::system_clock::now();
#endif

    // init value
    padDataClampToBorder(d_PaddedData, device_src_c_ptr, fftH, fftW, dataH, dataW, para.kernelH, para.kernelW, para.kernelY, para.kernelX);

#if TIMER
    auto t7 = std::chrono::system_clock::now();
#endif

    // make a FFT plan
    gpuErrorCheck(hipfftPlan2d(&fftPlanFwd, fftH, fftW, HIPFFT_R2C));
    gpuErrorCheck(hipfftPlan2d(&fftPlanInv, fftH, fftW, HIPFFT_C2R));

    // FFT data
    gpuErrorCheck(hipfftExecR2C(fftPlanFwd, (hipfftReal*)d_PaddedData, (hipfftComplex*)d_DataSpectrum));
    gpuErrorCheck(hipDeviceSynchronize());

#if TIMER
    auto t8 = std::chrono::system_clock::now();
#endif

    for (int i = 0; i < para.numberOfFilter; i++) {
        int kernel_offset = i * para.kernelH * para.kernelW;
        int data_offset = i * fftH * fftW;

        padKernel(d_PaddedKernel, &(d_Kernel[kernel_offset]), fftH, fftW, para.kernelH, para.kernelW, para.kernelY, para.kernelX);

        // FFT kernel
        gpuErrorCheck(hipfftExecR2C(fftPlanFwd, (hipfftReal*)d_PaddedKernel, (hipfftComplex*)d_KernelSpectrum));
        gpuErrorCheck(hipDeviceSynchronize());

        // mul
        modulateAndNormalize(d_TempSpectrum, d_DataSpectrum, d_KernelSpectrum, fftH, fftW, 1);
        gpuErrorCheck(hipfftExecC2R(fftPlanInv, (hipfftComplex*)d_TempSpectrum, (hipfftReal*)(&d_DepthResult[data_offset])));
        gpuErrorCheck(hipDeviceSynchronize());
    }

#if TIMER
    auto t9 = std::chrono::system_clock::now();
#endif

#if DEBUG 
    float* h_single;
    h_single = (float*)malloc(fftH * fftW * sizeof(float));
    for (int i = 0; i < para.numberOfFilter; i++) {
        int offs = i * fftH * fftW;
        gpuErrorCheck(hipMemcpy(h_single, &d_DepthResult[offs], fftH * fftW * sizeof(float), hipMemcpyDeviceToHost));
        displayImage(h_single, fftW, fftH, true);
    }
#endif

    cubeReduction(d_DepthResult, d_Result, fftH, fftW, dataH, dataW, depth);

#if TIMER
    auto t10 = std::chrono::system_clock::now();
#endif

    gpuErrorCheck(hipDeviceSynchronize());
    gpuErrorCheck(hipMemcpy(dst.data, d_Result, dataH * dataW * sizeof(uchar), hipMemcpyDeviceToHost));

#if TIMER
    auto t11 = std::chrono::system_clock::now();
#endif

    // free
    gpuErrorCheck(hipfftDestroy(fftPlanInv));
    gpuErrorCheck(hipfftDestroy(fftPlanFwd));
    gpuErrorCheck(hipFree(d_DataSpectrum));
    gpuErrorCheck(hipFree(d_KernelSpectrum));
    gpuErrorCheck(hipFree(d_PaddedData));
    gpuErrorCheck(hipFree(d_PaddedKernel));
    gpuErrorCheck(hipFree(d_TempSpectrum));
    gpuErrorCheck(hipFree(device_src_c_ptr));
    gpuErrorCheck(hipFree(d_Kernel));
    gpuErrorCheck(hipFree(d_DepthResult));
    gpuErrorCheck(hipDeviceReset());

#if TIMER
    auto t12 = std::chrono::system_clock::now();

    printTime(t1, t2, "source registering");
    printTime(t2, t3, "c channel extracting");
    printTime(t3, t4, "source unregistering");
    printTime(t4, t5, "get gabor filter");
    printTime(t5, t6, "hipMalloc");
    printTime(t6, t7, "padDataClampToBorder");
    printTime(t7, t8, "source FFT");
    printTime(t8, t9, "kernel FFT and mul");
    printTime(t9, t10, "CubeReduction");
    printTime(t10, t11, "D to H result");
    printTime(t11, t12, "free");
#endif
}

