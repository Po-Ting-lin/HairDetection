#include "hip/hip_runtime.h"
#include "hair_detection_kernel.cuh"
#include "utils.h"
#include <cuFFT.h>

#define TILE_DIM 32
#define BLOCK_DIM 8
#define EPSILON 1e-6

__global__ void extractLChannelWithInstrinicFunction(uchar* src, float* dst, int nx, int ny, int nz) {
    int x = threadIdx.x + TILE_DIM * blockIdx.x;
    int y = threadIdx.y + TILE_DIM * blockIdx.y;

    for (int i = 0; i < TILE_DIM; i += BLOCK_DIM) {
        // take pixel from DRAM
        uchar R = *(src + ((y + i) * nx * nz) + (x * nz) + 0);
        uchar G = *(src + ((y + i) * nx * nz) + (x * nz) + 1);
        uchar B = *(src + ((y + i) * nx * nz) + (x * nz) + 2);

        // RGB to XYZ
        float r = fdividef((float)R, 255.0f);
        float g = fdividef((float)G, 255.0f);
        float b = fdividef((float)B, 255.0f);
        r = ((r > 0.04045f) ? __powf(fdividef(r + 0.055f, 1.055f), 2.4f) : fdividef(r, 12.92f)) * 100.0f;
        g = ((g > 0.04045f) ? __powf(fdividef(g + 0.055f, 1.055f), 2.4f) : fdividef(g, 12.92f)) * 100.0f;
        b = ((b > 0.04045f) ? __powf(fdividef(b + 0.055f, 1.055f), 2.4f) : fdividef(b, 12.92f)) * 100.0f;

        // XYZ to LAB
        float Y = fdividef(0.2126f * r + 0.7152f * g + 0.0722f * b, 100.0f);
        Y = (Y > 0.008856f) ? cbrtf(Y) : fmaf(7.787f, Y, 0.1379f);
        float L = fmaf(116.0f, Y, -16.0f) * 2.55f;

        //printf("r: %d g: %d b: %d --- L: %f\n", R, G, B, L);

        // set pixel to DRAM
        *(dst + (y + i) * nx + x) = L;
    }
}

// very inefficient, frequently load from DRAM 
__global__ void entropyCalculationKernel(float* glcmA, float* glcmC, float* eA, float* eC, int dynamic) {
    int gid = threadIdx.x + blockDim.x * blockIdx.x;

    if (gid < dynamic) {
        float pA = 0.0f;
        float meanA = 0.0f;
        float entropyA = 0.0f;
        int tA = gid;
        // pA
        for (int r = 0; r < tA + 1; r++) {
            for (int c = 0; c < tA + 1; c++) {
                pA += *(glcmA + r * dynamic + c);
            }
        }

        // meanA
        for (int r = 0; r < tA + 1; r++) {
            for (int c = 0; c < tA + 1; c++) {
                meanA += ((float)r) * ((float)c) * (*(glcmA + r * dynamic + c));
            }
        }
        meanA /= pA;

        // entropyA
        for (int r = 0; r < tA + 1; r++) {
            for (int c = 0; c < tA + 1; c++) {
                float raw = (*(glcmA + r * dynamic + c));
                entropyA += ((float)r) * ((float)c) * raw * log2f((((float)r) * ((float)c) + EPSILON) / (meanA + EPSILON));
                entropyA += meanA * raw * log2f(meanA / (((float)r) + EPSILON) / (((float)c) + EPSILON) + EPSILON);
            }
        }

        eA[tA] = entropyA;
    }
    else {
        float pC = 0.0;
        float meanC = 0.0;
        float entropyC = 0.0f;
        int tC = gid - dynamic;
        // pC
        for (int r = tC + 1; r < dynamic; r++) {
            for (int c = tC + 1; c < dynamic; c++) {
                pC += *(glcmC + r * dynamic + c);
            }
        }

        // meanC
        for (int r = tC + 1; r < dynamic; r++) {
            for (int c = tC + 1; c < dynamic; c++) {
                meanC += ((float)r) * ((float)c) * (*(glcmC + r * dynamic + c));
            }
        }
        meanC /= pC;

        // entropyC
        for (int r = tC + 1; r < dynamic; r++) {
            for (int c = tC + 1; c < dynamic; c++) {
                float raw = (*(glcmC + r * dynamic + c));
                entropyC += ((float)r) * ((float)c) * raw * log2f((((float)r) * ((float)c) + EPSILON) / (meanC + EPSILON));
                entropyC += meanC * raw * log2f(meanC / (((float)r) + EPSILON) / (((float)c) + EPSILON) + EPSILON);
            }
        }

        eC[tC] = entropyC;
    }
}

void getHairMask(cv::Mat& src, cv::Mat& dst, HairDetectionParameters para) {
    auto t1 = std::chrono::system_clock::now();

    // declare 
    float
        * d_PaddedData,
        * d_Kernel,
        * d_PaddedKernel,
        * d_DepthResult;
    uchar
        * d_Result;

    fComplex
        * d_DataSpectrum,
        * d_KernelSpectrum,
        * d_TempSpectrum;

    hipfftHandle
        fftPlanFwd,
        fftPlanInv;

    uchar* src_ptr = src.data;
    const int dataH = src.rows;
    const int dataW = src.cols;
    const int depth = para.numberOfFilter;
    const int fftH = snapTransformSize(dataH + para.kernelH - 1);
    const int fftW = snapTransformSize(dataW + para.kernelW - 1);
    const unsigned long src_size = src.cols * src.rows * src.channels();
    const unsigned long src_byte_size = src_size * sizeof(uchar);
    const unsigned long src_c_size = src.cols * src.rows;
    const unsigned long src_c_byte_size = src_c_size * sizeof(float);

    // host data
    hipHostRegister(src_ptr, src_byte_size, hipHostRegisterDefault);

    auto t2 = std::chrono::system_clock::now();

    // device data
    uchar* device_src_ptr;
    float* device_src_c_ptr;
    gpuErrorCheck(hipMalloc((uchar**)&device_src_ptr, src_byte_size));
    gpuErrorCheck(hipMalloc((float**)&device_src_c_ptr, src_c_byte_size));

    // stream
    const int NUM_STREAMS = 6;
    int SRC_DATA_PER_STREAM = src_size / NUM_STREAMS;
    int DST_DATA_PER_STREAM = src_c_size / NUM_STREAMS;
    int SRC_BYTES_PER_STREAM = src_byte_size / NUM_STREAMS;
    int DST_BYTES_PER_STREAM = src_c_byte_size / NUM_STREAMS;

    hipStream_t stream[NUM_STREAMS];
    for (int i = 0; i < NUM_STREAMS; i++) {
        hipStreamCreate(&stream[i]);
    }

    int block_x_size = TILE_DIM;
    int block_y_size = BLOCK_DIM;
    int grid_x_size = (src.cols + TILE_DIM - 1) / TILE_DIM;
    int pruned_rows = src.rows / NUM_STREAMS;
    int grid_y_size = (pruned_rows + TILE_DIM - 1) / TILE_DIM;

    dim3 block(block_x_size, block_y_size);
    dim3 grid(grid_x_size, grid_y_size);

    int src_offset = 0;
    int dst_offset = 0;

    for (int i = 0; i < NUM_STREAMS; i++) {
        src_offset = i * SRC_DATA_PER_STREAM;
        dst_offset = i * DST_DATA_PER_STREAM;
        gpuErrorCheck(hipMemcpyAsync(&device_src_ptr[src_offset], &src_ptr[src_offset], SRC_BYTES_PER_STREAM, hipMemcpyHostToDevice, stream[i]));
        extractLChannelWithInstrinicFunction << < grid, block, 0, stream[i] >> > (&device_src_ptr[src_offset], &device_src_c_ptr[dst_offset], src.cols, pruned_rows, src.channels());
    }

    auto t3 = std::chrono::system_clock::now();

    gpuErrorCheck(hipDeviceSynchronize());

    for (int i = 0; i < NUM_STREAMS; i++) {
        hipStreamDestroy(stream[i]);
    }

    hipHostUnregister(src_ptr);
    gpuErrorCheck(hipFree(device_src_ptr));

    auto t4 = std::chrono::system_clock::now();

    // init data
    
    float* h_kernels = GaborFilterCube(para);

    auto t5 = std::chrono::system_clock::now();

    // allocation
    gpuErrorCheck(hipMalloc((void**)&d_Kernel, para.kernelH * para.kernelW * para.numberOfFilter * sizeof(float)));

    gpuErrorCheck(hipMalloc((void**)&d_PaddedData, fftH * fftW * sizeof(float)));
    gpuErrorCheck(hipMalloc((void**)&d_PaddedKernel, fftH * fftW * sizeof(float)));
    gpuErrorCheck(hipMalloc((void**)&d_DepthResult, fftH * fftW * para.numberOfFilter * sizeof(float)));
    gpuErrorCheck(hipMalloc((void**)&d_Result, dataH * dataW * sizeof(uchar)));

    gpuErrorCheck(hipMalloc((void**)&d_DataSpectrum, fftH * (fftW / 2 + 1) * sizeof(fComplex)));
    gpuErrorCheck(hipMalloc((void**)&d_KernelSpectrum, fftH * (fftW / 2 + 1) * sizeof(fComplex)));
    gpuErrorCheck(hipMalloc((void**)&d_TempSpectrum, fftH * (fftW / 2 + 1) * sizeof(fComplex)));

    // H to D
    gpuErrorCheck(hipMemcpy(d_Kernel, h_kernels, para.kernelH * para.kernelW * para.numberOfFilter * sizeof(float), hipMemcpyHostToDevice));

    auto t6 = std::chrono::system_clock::now();

    // init value
    padDataClampToBorder(d_PaddedData, device_src_c_ptr, fftH, fftW, dataH, dataW, para.kernelH, para.kernelW, para.kernelY, para.kernelX);


    auto t7 = std::chrono::system_clock::now();

    // make a FFT plan
    gpuErrorCheck(hipfftPlan2d(&fftPlanFwd, fftH, fftW, HIPFFT_R2C));
    gpuErrorCheck(hipfftPlan2d(&fftPlanInv, fftH, fftW, HIPFFT_C2R));

    // FFT data
    gpuErrorCheck(hipfftExecR2C(fftPlanFwd, (hipfftReal*)d_PaddedData, (hipfftComplex*)d_DataSpectrum));
    gpuErrorCheck(hipDeviceSynchronize());

    auto t8 = std::chrono::system_clock::now();

    for (int i = 0; i < para.numberOfFilter; i++) {
        int kernel_offset = i * para.kernelH * para.kernelW;
        int data_offset = i * fftH * fftW;

        padKernel(d_PaddedKernel, &(d_Kernel[kernel_offset]), fftH, fftW, para.kernelH, para.kernelW, para.kernelY, para.kernelX);

        // FFT kernel
        gpuErrorCheck(hipfftExecR2C(fftPlanFwd, (hipfftReal*)d_PaddedKernel, (hipfftComplex*)d_KernelSpectrum));
        gpuErrorCheck(hipDeviceSynchronize());

        // mul
        modulateAndNormalize(d_TempSpectrum, d_DataSpectrum, d_KernelSpectrum, fftH, fftW, 1);
        gpuErrorCheck(hipfftExecC2R(fftPlanInv, (hipfftComplex*)d_TempSpectrum, (hipfftReal*)(&d_DepthResult[data_offset])));
        gpuErrorCheck(hipDeviceSynchronize());
    }

    auto t9 = std::chrono::system_clock::now();

    // debug // 
    //float* h_single;
    //h_single = (float*)malloc(fftH * fftW * sizeof(float));

    //for (int i = 0; i < para.numberOfFilter; i++) {
    //    int offs = i * fftH * fftW;
    //    gpuErrorCheck(hipMemcpy(h_single, &d_DepthResult[offs], fftH * fftW * sizeof(float), hipMemcpyDeviceToHost));
    //    displayImage(h_single, fftW, fftH, true);
    //}
    ///////////

    CubeReduction(d_DepthResult, d_Result, fftH, fftW, dataH, dataW, depth);

    auto t10 = std::chrono::system_clock::now();

    gpuErrorCheck(hipDeviceSynchronize());
    gpuErrorCheck(hipMemcpy(dst.data, d_Result, dataH * dataW * sizeof(uchar), hipMemcpyDeviceToHost));

    auto t11 = std::chrono::system_clock::now();

    // free
    gpuErrorCheck(hipfftDestroy(fftPlanInv));
    gpuErrorCheck(hipfftDestroy(fftPlanFwd));

    gpuErrorCheck(hipFree(d_DataSpectrum));
    gpuErrorCheck(hipFree(d_KernelSpectrum));
    gpuErrorCheck(hipFree(d_PaddedData));
    gpuErrorCheck(hipFree(d_PaddedKernel));
    gpuErrorCheck(hipFree(d_TempSpectrum));
    gpuErrorCheck(hipFree(device_src_c_ptr));
    gpuErrorCheck(hipFree(d_Kernel));
    gpuErrorCheck(hipFree(d_DepthResult));

    gpuErrorCheck(hipDeviceReset());

    auto t12 = std::chrono::system_clock::now();

    //printTime(t1, t2, "source registering");
    //printTime(t2, t3, "c channel extracting");
    //printTime(t3, t4, "source unregistering");
    //printTime(t4, t5, "get gabor filter");
    //printTime(t5, t6, "hipMalloc");
    //printTime(t6, t7, "padDataClampToBorder");
    //printTime(t7, t8, "source FFT");
    //printTime(t8, t9, "kernel FFT and mul");
    //printTime(t9, t10, "CubeReduction");
    //printTime(t10, t11, "D to H result");
    //printTime(t11, t12, "free");
}



int entropyThesholdingGPU(cv::Mat& glcm) {
    int dynamic_range = 256;
    float
        * h_eA,
        * h_eC;
    float
        * d_glcmA,
        * d_glcmC,
        * d_eA,
        * d_eC;
    float* src_ptr = (float*)glcm.data;

    h_eA = (float*)malloc(dynamic_range * sizeof(float*));
    h_eC = (float*)malloc(dynamic_range * sizeof(float*));
    gpuErrorCheck(hipMalloc((void**)&d_glcmA, dynamic_range * dynamic_range * sizeof(float)));
    gpuErrorCheck(hipMalloc((void**)&d_glcmC, dynamic_range * dynamic_range * sizeof(float)));
    gpuErrorCheck(hipMalloc((void**)&d_eA, dynamic_range * sizeof(float)));
    gpuErrorCheck(hipMalloc((void**)&d_eC, dynamic_range * sizeof(float)));
    gpuErrorCheck(hipMemcpy(d_glcmA, src_ptr, dynamic_range * dynamic_range * sizeof(float), hipMemcpyHostToDevice));
    gpuErrorCheck(hipMemcpy(d_glcmC, d_glcmA, dynamic_range * dynamic_range * sizeof(float), hipMemcpyDeviceToDevice));

    dim3 block(TILE_DIM);
    dim3 grid(iDivUp(dynamic_range*2, TILE_DIM)); // 512 threads

    entropyCalculationKernel << <grid, block >> > (d_glcmA, d_glcmC, d_eA, d_eC, dynamic_range);
    gpuErrorCheck(hipDeviceSynchronize());

    gpuErrorCheck(hipMemcpy(h_eA, d_eA, dynamic_range * sizeof(float), hipMemcpyDeviceToHost));
    gpuErrorCheck(hipMemcpy(h_eC, d_eC, dynamic_range * sizeof(float), hipMemcpyDeviceToHost));

    int bestT = 0;
    float minLCM = FLT_MAX;

    for (int t = 0; t < dynamic_range; t++) {
        if (minLCM > h_eA[t] + h_eC[t]) {
            bestT = t;
            minLCM = h_eA[t] + h_eC[t];
        }
    }

    gpuErrorCheck(hipFree(d_glcmA));
    gpuErrorCheck(hipFree(d_glcmC));
    gpuErrorCheck(hipFree(d_eA));
    gpuErrorCheck(hipFree(d_eC));
    free(h_eA);
    free(h_eC);
    gpuErrorCheck(hipDeviceReset());

    return bestT;
}

void TestSumMatrix() {
    float
        * h_data,
        * h_test,
        * h_sum_matrix;
    float
        * d_data,
        * d_sum_matrix;

    float ref = 0;
    int raw_width = 32;
    int width = GetClosedWidth(raw_width);

    dim3 pre_sum_block(TILE_DIM, TILE_DIM);
    dim3 pre_sum_grid(iDivUp(raw_width, TILE_DIM), iDivUp(raw_width, TILE_DIM));
    dim3 sum_block(TILE_DIM, TILE_DIM);
    dim3 sum_grid(width / TILE_DIM, width / TILE_DIM);
    
    h_data = (float*)malloc(raw_width * raw_width * sizeof(float*));
    h_test = (float*)malloc(raw_width * raw_width * sizeof(float*));
    h_sum_matrix = (float*)malloc(sum_grid.x * sum_grid.x * sizeof(float*));

    for (int i = 0; i < raw_width * raw_width; i++) {
        h_data[i] = ((float)i / 100.0f);
        ref += ((float)i / 100.0f);
    }
    //Display2DArray(h_data, raw_width, raw_width);
    //std::cout << std::endl;
    
    gpuErrorCheck(hipMalloc((void**)&d_data, raw_width * raw_width * sizeof(float)));
    gpuErrorCheck(hipMalloc((void**)&d_sum_matrix, sum_grid.x * sum_grid.x * sizeof(float)));
    gpuErrorCheck(hipMemcpy(d_data, h_data, raw_width * raw_width * sizeof(float), hipMemcpyHostToDevice));

    // presum
    if (raw_width != width){
        PreSumXMatrix << <pre_sum_grid, pre_sum_block >> > (d_data, raw_width, raw_width, width);
        gpuErrorCheck(hipDeviceSynchronize());
        PreSumYMatrix << <pre_sum_grid, pre_sum_block >> > (d_data, raw_width, raw_width, width);
        gpuErrorCheck(hipDeviceSynchronize());
    }
    SumMatirx << <sum_grid, sum_block >> > (d_data, raw_width, width, d_sum_matrix);
    gpuErrorCheck(hipDeviceSynchronize());

    gpuErrorCheck(hipMemcpy(h_sum_matrix, d_sum_matrix, sum_grid.x * sum_grid.x * sizeof(float), hipMemcpyDeviceToHost));

    float result = 0.0f;
    for (int i = 0; i < sum_grid.x * sum_grid.x; i++) {
        result += h_sum_matrix[i];
    }

    printf("ref: %f, gpu: %f\n", ref, result);

    gpuErrorCheck(hipFree(d_data));
    gpuErrorCheck(hipFree(d_sum_matrix));
    free(h_data);
    free(h_sum_matrix);
    gpuErrorCheck(hipDeviceReset());
    return;
}

int GetClosedWidth(int width){
    int number = (int)log2(width);
    return pow(2, number);
}

__global__ void PreSumXMatrix(float* src, int nx, int raw_width, int new_width) {
    int x = threadIdx.x + blockDim.x * blockIdx.x;
    int y = threadIdx.y + blockDim.y * blockIdx.y;
    int diff = raw_width - new_width;
    if (x < raw_width && y < raw_width) {
        if ((x < new_width) && (x >= new_width - diff)) {
            src[y * nx + x] += src[y * nx + x + diff];
        }
    }
}

__global__ void PreSumYMatrix(float* src, int nx, int raw_width, int new_width) {
    int x = threadIdx.x + blockDim.x * blockIdx.x;
    int y = threadIdx.y + blockDim.y * blockIdx.y;
    int diff = raw_width - new_width;
    if (x < raw_width && y < raw_width) {
        if ((x < new_width && y < new_width) && (y >= new_width - diff)) {
            src[y * nx + x] += src[(y + diff) * nx + x];
        }
    }
}

__global__ void SumMatirx(float* src, int nx, int tx, float* sum) {
    __shared__ float smem[TILE_DIM * TILE_DIM];
    int x = threadIdx.x + blockDim.x * blockIdx.x;
    int y = threadIdx.y + blockDim.y * blockIdx.y;
    
    if (x < tx && y < tx) {
        smem[threadIdx.y * blockDim.x + threadIdx.x] = src[y * nx + x];
        for (int offx = blockDim.x / 2; offx > 0; offx /= 2) {
            if (threadIdx.x < offx) {
                smem[threadIdx.y * blockDim.x + threadIdx.x] += smem[threadIdx.y * blockDim.x + threadIdx.x + offx];

                __syncthreads();

                if (threadIdx.y < offx) {
                    smem[threadIdx.y * blockDim.x + threadIdx.x] += smem[(threadIdx.y + offx) * blockDim.x + threadIdx.x];
                }
            }
            __syncthreads();
        }
        if (threadIdx.x == 0 && threadIdx.y == 0) {
            sum[blockIdx.y * gridDim.x + blockIdx.x] = smem[threadIdx.y * blockDim.x + threadIdx.x];
            //printf("x: %d, y: %d -- %f\n", blockIdx.x, blockIdx.y, sum[blockIdx.y * gridDim.x + blockIdx.x]);
        }
    }
}

__global__ void SumSumAMatrix(float* sum_matrix, float* d_pA, int sum_matrix_size, int threshold) {
    __shared__ float smem[2 * TILE_DIM];
    int tid = threadIdx.x;

    // put the data in that block from DRAM to shared memory
    if (tid < sum_matrix_size) {
        smem[tid] = sum_matrix[tid];
    }
    else {
        smem[tid] = 0.0f;
    }
    __syncthreads(); // important

    // unrolling warp
    if (tid < 32) {
        volatile float* vsmem = smem;
        vsmem[tid] += vsmem[tid + 32]; 
        vsmem[tid] += vsmem[tid + 16];
        vsmem[tid] += vsmem[tid + 8];
        vsmem[tid] += vsmem[tid + 4];
        vsmem[tid] += vsmem[tid + 2];
        vsmem[tid] += vsmem[tid + 1];
    }

    if (tid == 0) {
        d_pA[threshold] = (float)(smem[0]);
    }
}

__global__ void SumSumMMatrix(float* sum_matrix, float *d_pA, float* d_mA, int sum_matrix_size, int threshold) {
    __shared__ float smem[2 * TILE_DIM];
    int tid = threadIdx.x;

    // put the data in that block from DRAM to shared memory
    if (tid < sum_matrix_size) {
        smem[tid] = sum_matrix[tid];
    }
    else {
        smem[tid] = 0.0f;
    }
    __syncthreads(); // important

    // unrolling warp
    if (tid < 32) {
        volatile float* vsmem = smem;
        vsmem[tid] += vsmem[tid + 32];
        vsmem[tid] += vsmem[tid + 16];
        vsmem[tid] += vsmem[tid + 8];
        vsmem[tid] += vsmem[tid + 4];
        vsmem[tid] += vsmem[tid + 2];
        vsmem[tid] += vsmem[tid + 1];
    }

    if (tid == 0) {
        //printf("threshold: %d, output: %f\n", threshold, (float)(smem[0]));
        d_mA[threshold] = (float)(smem[0] / d_pA[threshold]);
    }
}

__global__ void MultiplyRC(float* d_data_rc, float* d_data, int nx) {
    int x = threadIdx.x + blockDim.x * blockIdx.x;
    int y = threadIdx.y + blockDim.y * blockIdx.y;

    if (x < nx && y < nx) {
        d_data_rc[y * nx + x] = d_data[y * nx + x] * x * y;
    }
}

__global__ void ComputeEntropyMatrixKernel(float* d_data_computed, float* d_data, int nx, float* d_mA, int threshold) {
    int x = threadIdx.x + blockDim.x * blockIdx.x;
    int y = threadIdx.y + blockDim.y * blockIdx.y;

    if (x < nx && y < nx) {
        float meanA = d_mA[threshold];
        float p = d_data[y * nx + x];
        float value = p * x * y * log2f(((float)x * y + EPSILON) / (meanA + EPSILON));
        value += meanA * p * log2f(meanA / ((float)x + EPSILON) / ((float)y + EPSILON) + EPSILON);
        d_data_computed[y * nx + x] = value;
    }
}

void Test666() {
    float
        * h_data,
        * h_pA,
        * h_mA,
        * h_eA;

    float
        * d_data,
        * d_pA,
        * d_mA,
        * d_eA;

    int raw_width = 256;

    // init h_data
    h_data = (float*)malloc(raw_width * raw_width * sizeof(float*));
    h_pA = (float*)malloc(raw_width * sizeof(float*));
    h_mA = (float*)malloc(raw_width * sizeof(float*));
    h_eA = (float*)malloc(raw_width * sizeof(float*));
    for (int i = 0; i < raw_width * raw_width; i++) {
        h_data[i] = getRand() / 1000.0f;
    }
    //Display2DArray(h_data, raw_width, raw_width);
    //std::cout << std::endl;
    gpuErrorCheck(hipMalloc((void**)&d_data, raw_width * raw_width * sizeof(float)));
    gpuErrorCheck(hipMalloc((void**)&d_pA, raw_width * sizeof(float)));
    gpuErrorCheck(hipMalloc((void**)&d_mA, raw_width * sizeof(float)));
    gpuErrorCheck(hipMalloc((void**)&d_eA, raw_width * sizeof(float)));
    gpuErrorCheck(hipMemcpy(d_data, h_data, raw_width * raw_width * sizeof(float), hipMemcpyHostToDevice));

    GetPAArray(d_data, 256, d_pA);
    //gpuErrorCheck(hipMemcpy(h_pA, d_pA, raw_width * sizeof(float), hipMemcpyDeviceToHost));

    GetMAArray(d_data, 256, d_pA, d_mA);
    //gpuErrorCheck(hipMemcpy(h_mA, d_mA, raw_width * sizeof(float), hipMemcpyDeviceToHost));

    GetEAArray(d_data, 256, d_mA, d_eA);
    //gpuErrorCheck(hipMemcpy(h_eA, d_eA, raw_width * sizeof(float), hipMemcpyDeviceToHost));

    //// check pA
    //float* pA_ref;
    //pA_ref = (float*)malloc(raw_width * sizeof(float*));
    //for (int t = 0; t < raw_width; t++) {
    //    float pa_sum = 0.0f;
    //    for (int x = 0; x < t + 1; x++) {
    //        for (int y = 0; y < t + 1; y++) {
    //            pa_sum += h_data[y * raw_width + x];
    //        }
    //    }
    //    pA_ref[t] = pa_sum;
    //}
    //for (int i = 0; i < raw_width; i++) {
    //    //printf("ref: %f, gpu: %f\n", pA_ref[i], h_pA[i]);
    //}
    //
    //// check mA
    //float* mA_ref;
    //mA_ref = (float*)malloc(raw_width * sizeof(float*));
    //for (int t = 0; t < raw_width; t++) {
    //    float ma_sum = 0.0f;
    //    for (int x = 0; x < t + 1; x++) {
    //        for (int y = 0; y < t + 1; y++) {
    //            ma_sum += h_data[y * raw_width + x] * x * y;
    //        }
    //    }
    //    if (pA_ref[t] != 0.0f) {
    //        mA_ref[t] = ma_sum / pA_ref[t];
    //    }
    //    else {
    //        mA_ref[t] = 0.0f;
    //    }
    //}
    //for (int i = 0; i < raw_width; i++) {
    //    //printf("i: %d, ref: %f, gpu: %f\n", i, mA_ref[i], h_mA[i]);
    //}

    //// check eA
    //float* eA_ref;
    //eA_ref = (float*)malloc(raw_width * sizeof(float*));
    //for (int t = 0; t < raw_width; t++) {
    //    float ea_sum = 0.0f;
    //    float meanA = mA_ref[t];
    //    for (int x = 0; x < t + 1; x++) {
    //        for (int y = 0; y < t + 1; y++) {
    //            float p = h_data[y * raw_width + x];
    //            ea_sum += ((float)x) * ((float)y) * p * log2((((float)x) * ((float)y) + EPSILON) / (meanA + EPSILON));
    //            ea_sum += meanA * p * log2(meanA / (((float)x) + EPSILON) / (((float)y) + EPSILON) + EPSILON);
    //        }
    //    }
    //    eA_ref[t] = ea_sum;
    //}
    //for (int i = 0; i < raw_width; i++) {
    //    //printf("i: %d, ref: %f, gpu: %f\n", i, eA_ref[i], h_eA[i]);
    //}

    gpuErrorCheck(hipFree(d_data));
    free(h_data);
    gpuErrorCheck(hipDeviceReset());
    return;
}

int entropyThesholdingGPU2(cv::Mat& glcm) {
    int dynamic_range = 256;
    float
        * h_data,
        * h_reversed_data,
        * h_eA,
        * h_eC,
        * h_AC;

    float
        * d_data,
        * d_reversed_data,
        * d_pA,
        * d_mA,
        * d_eA,
        * d_pC,
        * d_mC,
        * d_eC;

    h_data = (float*)glcm.data;
    h_reversed_data = (float*)malloc(dynamic_range * dynamic_range * sizeof(float*));


    h_eA = (float*)malloc(dynamic_range * sizeof(float*));
    h_eC = (float*)malloc(dynamic_range * sizeof(float*));
    h_AC = (float*)malloc(dynamic_range * sizeof(float*));

    int j = dynamic_range * dynamic_range - 1;
    for (int i = 0; i < dynamic_range * dynamic_range; i++, j--) {
        h_reversed_data[j] = h_data[i];
    }

    gpuErrorCheck(hipMalloc((void**)&d_data, dynamic_range * dynamic_range * sizeof(float)));
    gpuErrorCheck(hipMalloc((void**)&d_reversed_data, dynamic_range * dynamic_range * sizeof(float)));
    gpuErrorCheck(hipMalloc((void**)&d_pA, dynamic_range * sizeof(float)));
    gpuErrorCheck(hipMalloc((void**)&d_mA, dynamic_range * sizeof(float)));
    gpuErrorCheck(hipMalloc((void**)&d_eA, dynamic_range * sizeof(float)));
    gpuErrorCheck(hipMalloc((void**)&d_pC, dynamic_range * sizeof(float)));
    gpuErrorCheck(hipMalloc((void**)&d_mC, dynamic_range * sizeof(float)));
    gpuErrorCheck(hipMalloc((void**)&d_eC, dynamic_range * sizeof(float)));
    gpuErrorCheck(hipMemcpy(d_data, h_data, dynamic_range * dynamic_range * sizeof(float), hipMemcpyHostToDevice));
    gpuErrorCheck(hipMemcpy(d_reversed_data, h_reversed_data, dynamic_range * dynamic_range * sizeof(float), hipMemcpyHostToDevice));

    GetPAArray(d_data, 256, d_pA);
    GetMAArray(d_data, 256, d_pA, d_mA);
    GetEAArray(d_data, 256, d_mA, d_eA);

    GetPAArray(d_reversed_data, 256, d_pC);
    GetMAArray(d_reversed_data, 256, d_pC, d_mC);
    GetEAArray(d_reversed_data, 256, d_mC, d_eC);

    gpuErrorCheck(hipMemcpy(h_eA, d_eA, dynamic_range * sizeof(float), hipMemcpyDeviceToHost));
    gpuErrorCheck(hipMemcpy(h_eC, d_eC, dynamic_range * sizeof(float), hipMemcpyDeviceToHost));

    int jj = dynamic_range - 1;
    for (int i = 0; i < dynamic_range; i++, jj--) {
        h_AC[i] = h_eA[i] + h_eC[jj];
        printf("i: %d, A:%f, C: %f, AC: %f\n", i, h_eA[i], h_eC[jj], h_AC[i]);
    }

    gpuErrorCheck(hipFree(d_data));
    gpuErrorCheck(hipFree(d_pA));
    gpuErrorCheck(hipFree(d_mA));
    gpuErrorCheck(hipFree(d_eA));
    free(h_eA);
    free(h_eC);
    free(h_AC);
    gpuErrorCheck(hipDeviceReset());
    return 0;
}

void GetPAArray(float* d_data, int full_width, float* d_pA) {
    float* d_buf;
    gpuErrorCheck(hipMalloc((void**)&d_buf, full_width * full_width * sizeof(float)));

    // from 32 to 255
    for (int i = TILE_DIM - 1; i < full_width; i++) {
        int raw_width = i + 1;
        int multiple_width = GetClosedWidth(raw_width);

        // refresh d_buf
        gpuErrorCheck(hipMemcpy(d_buf, d_data, full_width * full_width * sizeof(float), hipMemcpyDeviceToDevice));

        dim3 pre_sum_block(TILE_DIM, TILE_DIM);
        dim3 pre_sum_grid(iDivUp(raw_width, TILE_DIM), iDivUp(raw_width, TILE_DIM));
        dim3 sum_block(TILE_DIM, TILE_DIM);
        dim3 sum_grid(multiple_width / TILE_DIM, multiple_width / TILE_DIM);

        float* d_sum_matrix;
        gpuErrorCheck(hipMalloc((void**)&d_sum_matrix, sum_grid.x * sum_grid.x * sizeof(float)));
        gpuErrorCheck(hipMemset(d_sum_matrix, 0.0f, sum_grid.x * sum_grid.x * sizeof(float)));

        if (raw_width != multiple_width) {
            PreSumXMatrix << <pre_sum_grid, pre_sum_block >> > (d_buf, full_width, raw_width, multiple_width);
            gpuErrorCheck(hipDeviceSynchronize());
            PreSumYMatrix << <pre_sum_grid, pre_sum_block >> > (d_buf, full_width, raw_width, multiple_width);
            gpuErrorCheck(hipDeviceSynchronize());
        }
        SumMatirx << <sum_grid, sum_block >> > (d_buf, full_width, multiple_width, d_sum_matrix);
        gpuErrorCheck(hipDeviceSynchronize());

        SumSumAMatrix << <1, 2 * TILE_DIM >> > (d_sum_matrix, d_pA, sum_grid.x * sum_grid.x, i); // launch 64 threads to init smem
        gpuErrorCheck(hipDeviceSynchronize());

        // free
        gpuErrorCheck(hipFree(d_sum_matrix));
    }

    gpuErrorCheck(hipFree(d_buf));
}

void GetMAArray(float* d_data, int full_width, float* d_pA, float* d_mA) {
    float
        * d_buf,
        * d_data_rc;
    gpuErrorCheck(hipMalloc((void**)&d_buf, full_width * full_width * sizeof(float)));
    gpuErrorCheck(hipMalloc((void**)&d_data_rc, full_width * full_width * sizeof(float)));
    dim3 rc_block(TILE_DIM, TILE_DIM);
    dim3 rc_grid(iDivUp(full_width, TILE_DIM), iDivUp(full_width, TILE_DIM));
    MultiplyRC << <rc_grid, rc_block >> > (d_data_rc, d_data, full_width);
    gpuErrorCheck(hipDeviceSynchronize());

    // from 32 to 255
    for (int i = TILE_DIM - 1; i < full_width; i++) {
        int raw_width = i + 1;
        int multiple_width = GetClosedWidth(raw_width);

        // refresh d_buf
        gpuErrorCheck(hipMemcpy(d_buf, d_data_rc, full_width * full_width * sizeof(float), hipMemcpyDeviceToDevice));

        dim3 pre_sum_block(TILE_DIM, TILE_DIM);
        dim3 pre_sum_grid(iDivUp(raw_width, TILE_DIM), iDivUp(raw_width, TILE_DIM));
        dim3 sum_block(TILE_DIM, TILE_DIM);
        dim3 sum_grid(multiple_width / TILE_DIM, multiple_width / TILE_DIM);

        float* d_sum_matrix;
        gpuErrorCheck(hipMalloc((void**)&d_sum_matrix, sum_grid.x * sum_grid.x * sizeof(float)));
        gpuErrorCheck(hipMemset(d_sum_matrix, 0.0f, sum_grid.x * sum_grid.x * sizeof(float)));

        if (raw_width != multiple_width) {
            PreSumXMatrix << <pre_sum_grid, pre_sum_block >> > (d_buf, full_width, raw_width, multiple_width);
            gpuErrorCheck(hipDeviceSynchronize());
            PreSumYMatrix << <pre_sum_grid, pre_sum_block >> > (d_buf, full_width, raw_width, multiple_width);
            gpuErrorCheck(hipDeviceSynchronize());
        }
        SumMatirx << <sum_grid, sum_block >> > (d_buf, full_width, multiple_width, d_sum_matrix);
        gpuErrorCheck(hipDeviceSynchronize());
        SumSumMMatrix << <1, 2 * TILE_DIM >> > (d_sum_matrix, d_pA, d_mA, sum_grid.x * sum_grid.x, i);
        gpuErrorCheck(hipDeviceSynchronize());

        // free
        gpuErrorCheck(hipFree(d_sum_matrix));
    }

    gpuErrorCheck(hipFree(d_buf));
}

void GetEAArray(float* d_data, int full_width, float* d_mA, float* d_eA) {
    float
        * d_buf,
        * d_data_computed;
    gpuErrorCheck(hipMalloc((void**)&d_buf, full_width * full_width * sizeof(float)));
    gpuErrorCheck(hipMalloc((void**)&d_data_computed, full_width * full_width * sizeof(float)));

    // from 32 to 255
    for (int i = TILE_DIM - 1; i < full_width; i++) {
        int raw_width = i + 1;
        int multiple_width = GetClosedWidth(raw_width);

        // 
        dim3 rc_block(TILE_DIM, TILE_DIM);
        dim3 rc_grid(iDivUp(full_width, TILE_DIM), iDivUp(full_width, TILE_DIM));
        ComputeEntropyMatrixKernel << <rc_grid, rc_block >> > (d_data_computed, d_data, full_width, d_mA, i);
        gpuErrorCheck(hipDeviceSynchronize());
        //

        // refresh d_buf
        gpuErrorCheck(hipMemcpy(d_buf, d_data_computed, full_width * full_width * sizeof(float), hipMemcpyDeviceToDevice));

        dim3 pre_sum_block(TILE_DIM, TILE_DIM);
        dim3 pre_sum_grid(iDivUp(raw_width, TILE_DIM), iDivUp(raw_width, TILE_DIM));
        dim3 sum_block(TILE_DIM, TILE_DIM);
        dim3 sum_grid(multiple_width / TILE_DIM, multiple_width / TILE_DIM);

        float* d_sum_matrix;
        gpuErrorCheck(hipMalloc((void**)&d_sum_matrix, sum_grid.x * sum_grid.x * sizeof(float)));
        gpuErrorCheck(hipMemset(d_sum_matrix, 0.0f, sum_grid.x * sum_grid.x * sizeof(float)));

        if (raw_width != multiple_width) {
            PreSumXMatrix << <pre_sum_grid, pre_sum_block >> > (d_buf, full_width, raw_width, multiple_width);
            gpuErrorCheck(hipDeviceSynchronize());
            PreSumYMatrix << <pre_sum_grid, pre_sum_block >> > (d_buf, full_width, raw_width, multiple_width);
            gpuErrorCheck(hipDeviceSynchronize());
        }
        SumMatirx << <sum_grid, sum_block >> > (d_buf, full_width, multiple_width, d_sum_matrix);
        gpuErrorCheck(hipDeviceSynchronize());

        SumSumAMatrix << <1, 2 * TILE_DIM >> > (d_sum_matrix, d_eA, sum_grid.x * sum_grid.x, i);
        gpuErrorCheck(hipDeviceSynchronize());

        // free
        gpuErrorCheck(hipFree(d_sum_matrix));
    }

    gpuErrorCheck(hipFree(d_buf));
}